﻿#define COUNT 100

#include "hip/hip_runtime.h"

#include <string.h>
#include <stdio.h>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>


unsigned int Rand(unsigned int randx)
{
    randx = randx * 1103515245 + 12345;
    return randx & 2147483647;
}


 int* random(int *c)
{
    int i = 0;
    c[i] = 100;
    for (i = 0;i < COUNT;i++) {
        c[i+1] = Rand(c[i]); 
    }
    return c;
}

int main()
{
    /*int* c;
    cudaMalloc(&c, COUNT * sizeof(int));*/
    int *c;
    int i;
    c = (int*)malloc(COUNT * sizeof(int));
    c = random(c);
    for (i = 0;i < COUNT;i++) {
        std::cout << "rand : " << float(c[i])/ 2147483647 << "\n";
    }
}

// Helper function for using CUDA to add vectors in parallel.

